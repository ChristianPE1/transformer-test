// cuda_utils.cu
#include "utils/cuda_utils.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void* allocateCudaMemory(size_t size) {
    void* d_ptr;
    checkCudaError(hipMalloc(&d_ptr, size), "Failed to allocate CUDA memory");
    return d_ptr;
}

void copyToDevice(void* d_ptr, const void* h_ptr, size_t size) {
    checkCudaError(hipMemcpy(d_ptr, h_ptr, size, hipMemcpyHostToDevice), "Failed to copy to device");
}

void copyToHost(void* h_ptr, const void* d_ptr, size_t size) {
    checkCudaError(hipMemcpy(h_ptr, d_ptr, size, hipMemcpyDeviceToHost), "Failed to copy to host");
}

void freeCudaMemory(void* d_ptr) {
    checkCudaError(hipFree(d_ptr), "Failed to free CUDA memory");
}

void synchronizeCuda() {
    checkCudaError(hipDeviceSynchronize(), "CUDA device synchronization failed");
}