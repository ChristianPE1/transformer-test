#include "hip/hip_runtime.h"
#include "utils/matrix.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <stdexcept>
#include <cstdio>
#include <vector>
#include <algorithm>

__global__ void matrixAddKernel(float *a, float *b, float *result, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        result[idx] = a[idx] + b[idx];
    }
}

Matrix::Matrix(int rows, int cols) : rows(rows), cols(cols), on_device(true)
{
    hipMalloc(&data, rows * cols * sizeof(float));
    hipMemset(data, 0, rows * cols * sizeof(float));
}

Matrix::Matrix(int rows, int cols, float init_val) : rows(rows), cols(cols), on_device(true)
{
    hipMalloc(&data, rows * cols * sizeof(float));

    std::vector<float> host_data(rows * cols, init_val);
    hipMemcpy(data, host_data.data(), rows * cols * sizeof(float), hipMemcpyHostToDevice);
}

Matrix::~Matrix()
{
    if (data && on_device)
    {
        hipFree(data);
    }
}

Matrix::Matrix(const Matrix &other) : rows(other.rows), cols(other.cols), on_device(true)
{
    hipMalloc(&data, rows * cols * sizeof(float));
    hipMemcpy(data, other.data, rows * cols * sizeof(float), hipMemcpyDeviceToDevice);
}

Matrix &Matrix::operator=(const Matrix &other)
{
    if (this != &other)
    {
        if (data && on_device)
        {
            hipFree(data);
        }

        rows = other.rows;
        cols = other.cols;
        on_device = true;

        hipMalloc(&data, rows * cols * sizeof(float));
        hipMemcpy(data, other.data, rows * cols * sizeof(float), hipMemcpyDeviceToDevice);
    }
    return *this;
}

Matrix Matrix::add(const Matrix &other) const
{
    if (rows != other.rows || cols != other.cols)
    {
        throw std::runtime_error("Matrix dimensions don't match for addition");
    }

    Matrix result(rows, cols);
    int size = rows * cols;

    // Use CPU implementation for stability
    std::vector<float> host_a(size);
    std::vector<float> host_b(size);
    std::vector<float> host_result(size);
    
    // Copy data to host
    hipMemcpy(host_a.data(), data, size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(host_b.data(), other.data, size * sizeof(float), hipMemcpyDeviceToHost);
    
    // Perform addition on CPU
    for (int i = 0; i < size; i++) {
        host_result[i] = host_a[i] + host_b[i];
    }
    
    // Copy result back to GPU
    hipMemcpy(result.data, host_result.data(), size * sizeof(float), hipMemcpyHostToDevice);

    return result;
}

Matrix Matrix::multiply(const Matrix &other) const
{
    if (cols != other.rows)
    {
        throw std::runtime_error("Matrix dimensions don't match for multiplication");
    }

    Matrix result(rows, other.cols);
    
    // Use CPU implementation for stability
    std::vector<float> host_a(rows * cols);
    std::vector<float> host_b(other.rows * other.cols);
    std::vector<float> host_result(rows * other.cols, 0.0f);
    
    // Copy data to host
    hipMemcpy(host_a.data(), data, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(host_b.data(), other.data, other.rows * other.cols * sizeof(float), hipMemcpyDeviceToHost);
    
    // Perform matrix multiplication on CPU
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < other.cols; j++) {
            float sum = 0.0f;
            for (int k = 0; k < cols; k++) {
                sum += host_a[i * cols + k] * host_b[k * other.cols + j];
            }
            host_result[i * other.cols + j] = sum;
        }
    }
    
    // Copy result back to GPU
    hipMemcpy(result.data, host_result.data(), rows * other.cols * sizeof(float), hipMemcpyHostToDevice);
    
    return result;
}

void Matrix::copyFromHost(const std::vector<float> &hostData)
{
    if (hostData.size() != rows * cols)
    {
        throw std::runtime_error("Host data size doesn't match matrix size");
    }

    hipMemcpy(data, hostData.data(), rows * cols * sizeof(float), hipMemcpyHostToDevice);
}

void Matrix::copyToHost(std::vector<float> &hostData) const
{
    hostData.resize(rows * cols);
    hipMemcpy(hostData.data(), data, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
}

float Matrix::getElement(int row, int col) const
{
    float value;
    hipMemcpy(&value, &data[row * cols + col], sizeof(float), hipMemcpyDeviceToHost);
    return value;
}

void Matrix::setElement(int row, int col, float value)
{
    hipMemcpy(&data[row * cols + col], &value, sizeof(float), hipMemcpyHostToDevice);
}