#include "hip/hip_runtime.h"
// filepath: cuda-transformer/cuda-transformer/src/layers/feed_forward.cu
#include "feed_forward.cuh"
#include "utils/cuda_utils.cuh"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <cmath>
#include <cstdlib> // Para rand()
#include <ctime>   // Para time()
#include <iostream>

__global__ void feedForwardKernel(
    const float* input, float* output,
    const float* W1, const float* b1,
    const float* W2, const float* b2,
    int rows, int input_dim, int d_ff, int output_dim)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows) {
        // First layer: Linear + ReLU
        for (int j = 0; j < d_ff; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < input_dim; ++k) {
                sum += input[idx * input_dim + k] * W1[k * d_ff + j];
            }
            sum += b1[j];
            output[idx * d_ff + j] = fmaxf(0.0f, sum); // ReLU
        }

        // Second layer: Linear
        for (int j = 0; j < output_dim; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < d_ff; ++k) {
                sum += output[idx * d_ff + k] * W2[k * output_dim + j];
            }
            sum += b2[j];
            output[idx * output_dim + j] = sum; // No activation
        }
    }
}

Matrix FeedForward::forward(const Matrix &input) {
    int rows = input.getRows();
    int input_dim = input.getCols();
    int d_ff = this->d_ff;
    int output_dim = this->d_model;

    // O aqui CPU para mas estabilidad
    std::vector<float> input_h, W1_h, W2_h, b1_h, b2_h;
    input.copyToHost(input_h);
    W1.copyToHost(W1_h);
    W2.copyToHost(W2_h);
    
    b1_h.resize(d_ff);
    b2_h.resize(d_model);
    hipMemcpy(b1_h.data(), b1, d_ff * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b2_h.data(), b2, d_model * sizeof(float), hipMemcpyDeviceToHost);

    std::vector<float> output_h(rows * output_dim);
    
    // Create intermediate activation matrix for the hidden layer
    std::vector<float> hidden(rows * d_ff);
    
    // First layer: input -> hidden (with ReLU)
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < d_ff; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < input_dim; ++k) {
                sum += input_h[i * input_dim + k] * W1_h[k * d_ff + j];
            }
            sum += b1_h[j];
            hidden[i * d_ff + j] = fmaxf(0.0f, sum); // ReLU activation
        }
    }
    
    // Second layer: hidden -> output (no activation)
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < output_dim; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < d_ff; ++k) {
                sum += hidden[i * d_ff + k] * W2_h[k * output_dim + j];
            }
            sum += b2_h[j];
            output_h[i * output_dim + j] = sum;
        }
    }
    
    Matrix output(rows, output_dim);
    output.copyFromHost(output_h);
    
    return output;
}

FeedForward::FeedForward(size_t d_model, size_t d_ff) 
    : d_model(d_model), d_ff(d_ff), W1(d_model, d_ff), W2(d_ff, d_model),
      grad_W1(d_model, d_ff, 0.0f), grad_W2(d_ff, d_model, 0.0f) {
    // Alocar memoria para b1 y b2
    hipMalloc(&b1, d_ff * sizeof(float));
    hipMalloc(&b2, d_model * sizeof(float));
    
    // Alocar memoria para gradientes de biases
    hipMalloc(&grad_b1, d_ff * sizeof(float));
    hipMalloc(&grad_b2, d_model * sizeof(float));
    
    // Inicializar gradientes de biases a cero
    hipMemset(grad_b1, 0, d_ff * sizeof(float));
    hipMemset(grad_b2, 0, d_model * sizeof(float));
    
    initializeWeights();
}

FeedForward::~FeedForward() {
    if (b1) hipFree(b1);
    if (b2) hipFree(b2);
    if (grad_b1) hipFree(grad_b1);
    if (grad_b2) hipFree(grad_b2);
}

__global__ void initializeWeightsKernel(float* weights, int size, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);
        weights[idx] = hiprand_normal(&state) * 0.1f; // Xavier-like initialization
    }
}

void FeedForward::initializeWeights() {
    // Use proper seed
    srand(static_cast<unsigned>(time(nullptr)));
    
    // Initialize W1 weights with Xavier initialization
    int W1_size = d_model * d_ff;
    std::vector<float> W1_data(W1_size);
    float xavier_w1 = sqrt(2.0f / (d_model + d_ff)); // Xavier initialization
    for (int i = 0; i < W1_size; ++i) {
        W1_data[i] = ((float)rand() / RAND_MAX - 0.5f) * 2.0f * xavier_w1;
    }
    W1.copyFromHost(W1_data);

    // Initialize W2 weights with Xavier initialization
    int W2_size = d_ff * d_model;
    std::vector<float> W2_data(W2_size);
    float xavier_w2 = sqrt(2.0f / (d_ff + d_model)); // Xavier initialization
    for (int i = 0; i < W2_size; ++i) {
        W2_data[i] = ((float)rand() / RAND_MAX - 0.5f) * 2.0f * xavier_w2;
    }
    W2.copyFromHost(W2_data);

    // Initialize biases to zero
    std::vector<float> b1_data(d_ff, 0.0f);
    std::vector<float> b2_data(d_model, 0.0f);
    hipMemcpy(b1, b1_data.data(), d_ff * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b2, b2_data.data(), d_model * sizeof(float), hipMemcpyHostToDevice);
    
    std::cout << "[FEEDFORWARD] Weights initialized with Xavier initialization" << std::endl;
    std::cout << "[FEEDFORWARD] W1 scale: " << xavier_w1 << ", W2 scale: " << xavier_w2 << std::endl;
}

Matrix FeedForward::backward(const Matrix &grad_output, const Matrix &input) {
    int batch_size = grad_output.getRows();
    int output_dim = grad_output.getCols();
    int input_dim = input.getCols();
    
    // Initialize gradient for input
    Matrix grad_input(batch_size, input_dim, 0.0f);
    
    // FFN: output = W2 * ReLU(W1 * input + b1) + b2
    // We need to compute gradients through ReLU activation
    
    std::vector<float> h_grad_output, h_input;
    grad_output.copyToHost(h_grad_output);
    input.copyToHost(h_input);
    
    // Get current weights for gradient computation
    std::vector<float> W1_data, W2_data;
    W1.copyToHost(W1_data);
    W2.copyToHost(W2_data);
    
    // Initialize gradient accumulators
    std::vector<float> grad_W1_data(d_model * d_ff, 0.0f);
    std::vector<float> grad_W2_data(d_ff * d_model, 0.0f);
    std::vector<float> grad_b1_h(d_ff, 0.0f);
    std::vector<float> grad_b2_h(d_model, 0.0f);
    std::vector<float> grad_input_h(batch_size * input_dim, 0.0f);
    
    // REAL BACKWARD COMPUTATION
    for (int b = 0; b < batch_size; ++b) {
        // Step 1: Compute intermediate values (W1 * input + b1)
        std::vector<float> z1(d_ff, 0.0f);  // Before ReLU
        std::vector<float> a1(d_ff, 0.0f);  // After ReLU
        
        for (int j = 0; j < d_ff; ++j) {
            for (int i = 0; i < input_dim; ++i) {
                z1[j] += W1_data[i * d_ff + j] * h_input[b * input_dim + i];
            }
            // Add bias (simplified - would get from device in real implementation)
            a1[j] = fmaxf(0.0f, z1[j]); // ReLU activation
        }
        
        // Step 2: Gradient of loss w.r.t b2 = grad_output
        for (int i = 0; i < d_model; ++i) {
            grad_b2_h[i] += h_grad_output[b * d_model + i];
        }
        
        // Step 3: Gradient of loss w.r.t W2
        for (int i = 0; i < d_model; ++i) {
            for (int j = 0; j < d_ff; ++j) {
                grad_W2_data[j * d_model + i] += a1[j] * h_grad_output[b * d_model + i];
            }
        }
        
        // Step 4: Gradient of loss w.r.t a1 (intermediate activation)
        std::vector<float> grad_a1(d_ff, 0.0f);
        for (int j = 0; j < d_ff; ++j) {
            for (int i = 0; i < d_model; ++i) {
                grad_a1[j] += W2_data[j * d_model + i] * h_grad_output[b * d_model + i];
            }
        }
        
        // Step 5: Gradient through ReLU (derivative is 1 if z1 > 0, else 0)
        std::vector<float> grad_z1(d_ff, 0.0f);
        for (int j = 0; j < d_ff; ++j) {
            grad_z1[j] = (z1[j] > 0.0f) ? grad_a1[j] : 0.0f;
        }
        
        // Step 6: Gradient w.r.t b1
        for (int j = 0; j < d_ff; ++j) {
            grad_b1_h[j] += grad_z1[j];
        }
        
        // Step 7: Gradient w.r.t W1
        for (int i = 0; i < input_dim; ++i) {
            for (int j = 0; j < d_ff; ++j) {
                grad_W1_data[i * d_ff + j] += h_input[b * input_dim + i] * grad_z1[j];
            }
        }
        
        // Step 8: Gradient w.r.t input
        for (int i = 0; i < input_dim; ++i) {
            for (int j = 0; j < d_ff; ++j) {
                grad_input_h[b * input_dim + i] += W1_data[i * d_ff + j] * grad_z1[j];
            }
        }
    }
    
    // Store gradients for weight updates
    grad_W1.copyFromHost(grad_W1_data);
    grad_W2.copyFromHost(grad_W2_data);
    
    hipMemcpy(grad_b1, grad_b1_h.data(), d_ff * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(grad_b2, grad_b2_h.data(), d_model * sizeof(float), hipMemcpyHostToDevice);
    
    grad_input.copyFromHost(grad_input_h);
    
    std::cout << "[FEEDFORWARD] Real backward pass completed - gradients computed for W1, W2, b1, b2" << std::endl;
    
    return grad_input;
}

void FeedForward::updateWeights(float learning_rate) {
    if (learning_rate == 0.0f) {
        std::cout << "[FEEDFORWARD] WARNING: Learning rate is 0!" << std::endl;
        return;
    }
    
    // Update W1
    std::vector<float> W1_data, grad_W1_data;
    W1.copyToHost(W1_data);
    grad_W1.copyToHost(grad_W1_data);
    
    // Check for NaN or inf in weights and gradients
    bool has_nan_w1 = false, has_nan_grad_w1 = false;
    for (size_t i = 0; i < W1_data.size(); ++i) {
        if (std::isnan(W1_data[i]) || std::isinf(W1_data[i])) has_nan_w1 = true;
        if (std::isnan(grad_W1_data[i]) || std::isinf(grad_W1_data[i])) has_nan_grad_w1 = true;
    }
    
    if (has_nan_w1 || has_nan_grad_w1) {
        std::cout << "[FEEDFORWARD] ERROR: NaN/Inf detected in W1 weights or gradients! Reinitializing..." << std::endl;
        // Reinitialize W1
        for (size_t i = 0; i < W1_data.size(); ++i) {
            W1_data[i] = ((float)rand() / RAND_MAX - 0.5f) * 0.1f; // Smaller initialization
        }
        W1.copyFromHost(W1_data);
        // Skip this update
        return;
    }
    
    // Clip gradients para acelerar - menos conservador
    for (size_t i = 0; i < grad_W1_data.size(); ++i) {
        if (grad_W1_data[i] > 2.0f) grad_W1_data[i] = 2.0f;
        if (grad_W1_data[i] < -2.0f) grad_W1_data[i] = -2.0f;
    }
    
    for (size_t i = 0; i < W1_data.size(); ++i) {
        W1_data[i] -= learning_rate * grad_W1_data[i];
        
        // RELAJAR: Limitar pesos W1 para acelerar pero mantener estabilidad
        if (W1_data[i] > 1.0f) W1_data[i] = 1.0f;
        if (W1_data[i] < -1.0f) W1_data[i] = -1.0f;
    }
    W1.copyFromHost(W1_data);
    
    // Update W2
    std::vector<float> W2_data, grad_W2_data;
    W2.copyToHost(W2_data);
    grad_W2.copyToHost(grad_W2_data);
    
    // Check for NaN or inf in W2
    bool has_nan_w2 = false, has_nan_grad_w2 = false;
    for (size_t i = 0; i < W2_data.size(); ++i) {
        if (std::isnan(W2_data[i]) || std::isinf(W2_data[i])) has_nan_w2 = true;
        if (std::isnan(grad_W2_data[i]) || std::isinf(grad_W2_data[i])) has_nan_grad_w2 = true;
    }
    
    if (has_nan_w2 || has_nan_grad_w2) {
        std::cout << "[FEEDFORWARD] ERROR: NaN/Inf detected in W2 weights or gradients! Reinitializing..." << std::endl;
        // Reinitialize W2
        for (size_t i = 0; i < W2_data.size(); ++i) {
            W2_data[i] = ((float)rand() / RAND_MAX - 0.5f) * 0.1f; // Smaller initialization
        }
        W2.copyFromHost(W2_data);
        // Skip this update
        return;
    }
    
    // Clip gradients para acelerar
    for (size_t i = 0; i < grad_W2_data.size(); ++i) {
        if (grad_W2_data[i] > 2.0f) grad_W2_data[i] = 2.0f;
        if (grad_W2_data[i] < -2.0f) grad_W2_data[i] = -2.0f;
    }
    
    for (size_t i = 0; i < W2_data.size(); ++i) {
        W2_data[i] -= learning_rate * grad_W2_data[i];
        
        // RELAJAR: Limitar pesos W2 para acelerar
        if (W2_data[i] > 1.0f) W2_data[i] = 1.0f;
        if (W2_data[i] < -1.0f) W2_data[i] = -1.0f;
    }
    W2.copyFromHost(W2_data);
    
    // Update biases (simplified - would use CUDA kernels in optimized version)
    std::vector<float> b1_h(d_ff), b2_h(d_model);
    std::vector<float> grad_b1_h(d_ff), grad_b2_h(d_model);
    
    hipMemcpy(b1_h.data(), b1, d_ff * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b2_h.data(), b2, d_model * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(grad_b1_h.data(), grad_b1, d_ff * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(grad_b2_h.data(), grad_b2, d_model * sizeof(float), hipMemcpyDeviceToHost);
    
    // Clip bias gradients and check for NaN
    for (size_t i = 0; i < d_ff; ++i) {
        if (std::isnan(grad_b1_h[i]) || std::isinf(grad_b1_h[i])) {
            grad_b1_h[i] = 0.0f;
        } else {
            if (grad_b1_h[i] > 2.0f) grad_b1_h[i] = 2.0f;
            if (grad_b1_h[i] < -2.0f) grad_b1_h[i] = -2.0f;
        }
        b1_h[i] -= learning_rate * grad_b1_h[i];
        
        // Relajar bias b1
        if (b1_h[i] > 1.0f) b1_h[i] = 1.0f;
        if (b1_h[i] < -1.0f) b1_h[i] = -1.0f;
    }
    
    for (size_t i = 0; i < d_model; ++i) {
        if (std::isnan(grad_b2_h[i]) || std::isinf(grad_b2_h[i])) {
            grad_b2_h[i] = 0.0f;
        } else {
            if (grad_b2_h[i] > 2.0f) grad_b2_h[i] = 2.0f;
            if (grad_b2_h[i] < -2.0f) grad_b2_h[i] = -2.0f;
        }
        b2_h[i] -= learning_rate * grad_b2_h[i];
        
        // Relajar bias b2
        if (b2_h[i] > 1.0f) b2_h[i] = 1.0f;
        if (b2_h[i] < -1.0f) b2_h[i] = -1.0f;
    }
    
    hipMemcpy(b1, b1_h.data(), d_ff * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b2, b2_h.data(), d_model * sizeof(float), hipMemcpyHostToDevice);
    
    // Reset gradients
    grad_W1 = Matrix(d_model, d_ff, 0.0f);
    grad_W2 = Matrix(d_ff, d_model, 0.0f);
    hipMemset(grad_b1, 0, d_ff * sizeof(float));
    hipMemset(grad_b2, 0, d_model * sizeof(float));
    
    std::cout << "[FEEDFORWARD] Weights updated with lr=" << learning_rate << std::endl;
}